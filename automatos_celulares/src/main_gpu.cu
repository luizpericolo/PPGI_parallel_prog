#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void generate_map(hiprandState* devState, int n_maps, int* grid, int width, int height);
__global__ void setup_rnd_kernel (hiprandState* state, unsigned long seed);

__device__ void apply_cave_generation_rule();
__device__ void create_random_initial_population(hiprandState* devState, int* grid, int width, int height, int fill_percent);
__device__ int rand(hiprandState* localRand, int max);

__device__ void pretty_print_map(int* map, int width, int height);

__global__ void generate_map(hiprandState* devState, int n_maps, int* grid, int width, int height)
{
	int i;

	/* Literature says so... */
	int n_iterations = 5;

	/*
		This is the kernel function
	*/

	create_random_initial_population(devState, grid, width, height, 45);

	pretty_print_map(grid, width, height);

	/*pthread_params *params = (pthread_params*) thread_params;

	int n_elements = CELL_COUNT / N_THREADS;

	int start = n_elements * (params->id);
	int end = start + n_elements;

	apply_cave_generation_rule(current_gen, start, end);
	
	free(params);

	pthread_exit(NULL);*/

	

	

//	printf("[BlockId]: %d, [ThreadId]: %d\n", blockIdx.x, threadIdx.x);

}

__global__ void setup_rnd_kernel (hiprandState* state, unsigned long seed)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ void apply_cave_generation_rule()
{
   /* int i;

    int n_count = 0;

    for(i=startPos; i<endPos; i++)
    {
        n_count = count_neighbors(grid, i);

        // Applying the B678 rule.
        if(grid[i] == 0)
        {
            if((n_count == 6) || (n_count == 7) || (n_count == 8))
                next_gen[i] = 1;
            else
                next_gen[i] = grid[i];
        }

        // Applying the S345678
        if(grid[i] == 1)
        {
            if((n_count == 0) || (n_count == 1) || (n_count == 2))
                next_gen[i] = 0;
            else
                next_gen[i] = grid[i];
        }
    }*/
}

__device__ void create_random_initial_population(hiprandState* devState, int* grid, int width, int height, int fill_percent)
{
	/* This function creates a grid with a random distribution of 1 and 0 cells given width, height and a fill_percentage. */
    int i = 0;

    int count = 0;

    int cell_count = width * height;

    for(i=0; i<cell_count; i++)
    {
        if(rand(devState, 100) < fill_percent)
        {
            grid[i] = 1;
            count++;
        }
        else
        {
            grid[i] = 0;
        }
            
    }

}

__device__ int rand(hiprandState* localRand, int max) 
{
    int ind = threadIdx.x;
    hiprandState localState = localRand[ind];
    float rnd = hiprand_uniform( &localState );
    localRand[ind] = localState;
    
    return int(rnd * max);
}

__device__ void pretty_print_map(int* map, int width, int height)
{
	int i;

	for(i=0; i<width*height; i++)
	{
		if(i%width == 0)
			printf("\n");
		
		if(map[i] == 0)
			printf(".");

		if(map[i] == 1)
			printf("@");
	}

	printf("\n");
}

int main(int argc, char* argv[])
{
	
	int *cudaGrid, *grid;
	hiprandState* devStates;

    srand(time(NULL));
	
	/*int i, n, t, s;

	struct timeval inicio, fim;
    tsc_counter tsc1, tsc2;
    long long unsigned int clock;
    double tempo, tempo_total;*/

    int N_THREADS, N_BLOCKS;

    N_THREADS = 1;
    N_BLOCKS = 1;

    int n_maps = 5;

    int width, height;

    width = 10;
    height = 10;

    hipMalloc(&devStates, N_THREADS * N_BLOCKS * sizeof( hiprandState ));
  	setup_rnd_kernel <<<N_THREADS, N_BLOCKS>>> ( devStates, time(NULL) );


	if (hipMalloc (&cudaGrid, sizeof(int) * width * height) != hipSuccess) 
		{ printf("Erro cudaMalloc\n"); return -1; }

	grid = (int*) malloc(width * height * sizeof(int));

    generate_map <<<N_THREADS, N_BLOCKS>>>(devStates, n_maps, cudaGrid, width, height);
    hipDeviceSynchronize();

    if (hipMemcpy(grid, cudaGrid, sizeof(int) * width*height, hipMemcpyDeviceToHost) != hipSuccess) 
       { printf("Erro cudaMemcpy\n"); return -1; }

    printf("Depois de chamar!\n");

    int i;
    for(i=0; i<width*height; i++)
	{
		if(i%width == 0)
			printf("\n");
		
		if(grid[i] == 0)
			printf(".");

		if(grid[i] == 1)
			printf("@");
	}

	printf("\n");

    /*for(s=0; s<N_SIM; s++)
    {
        gettimeofday(&inicio, NULL);
        RDTSC(tsc1); 

        for(i=0; i<N_MAPS; i++)
        {
            current_gen = create_random_initial_population();
        	next_gen = malloc(CELL_COUNT * sizeof(int));
        	fill_borders(current_gen);

        	for(n=0; n<N_ITER; n++)
        	{
        		for(t=0; t<N_THREADS; t++)
        		{
        			pthread_params *params;
        			params = malloc(sizeof(pthread_params));
        			params->id = t;

        			pthread_create(&tid[t], NULL,
                        transition_cells, (void*) params);
        		}

        		for(t=0; t<N_THREADS; t++)
        		{
        			pthread_join(tid[t], NULL);

                }

        		current_gen = next_gen;
        		fill_borders(current_gen);
                //print_grid(current_gen);
        		
        	}

            //printf("Mapa #%d:\n", i+1);
            //print_grid(current_gen);
            free(current_gen);
            //free(next_gen);
        }

        RDTSC(tsc2);
        gettimeofday(&fim, NULL);

        printf("Run #%d\n", s);
        tempo = (fim.tv_sec - inicio.tv_sec) * 1000 + (fim.tv_sec - inicio.tv_sec)/1000;
        tempo_total += tempo;
        printf("Tempo: %.2lf\n", tempo);

        clock = tsc2.int64 - tsc1.int64;
        printf("Tempo: %.2lf(ms) Clocks: %.2e\n", tempo/N_MAPS, (double)clock/N_ITER);
        printf("Clock/tempo: %.2e\n\n", clock/tempo);
    }

    printf("Tempo Total: %.2lf\n", tempo_total);
    printf("Tempo médio: %.2lf\n", (double) tempo_total/N_SIM);
	
	pthread_exit(NULL);*/

	return 0;
}